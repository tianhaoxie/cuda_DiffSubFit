#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <vector>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <stdio.h>
#include <sstream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <chrono>


__global__ void sorted_adjacency_list(int* f,int N_f,int* adj,const int vertex_num,const int num_neighbor){
    int present_thread = blockIdx.x * blockDim.x + threadIdx.x;
    if (present_thread>vertex_num-1){
        return;
    }
    int vert_idx = present_thread;
    int face_idx;
    int p_in;
    int tmp[30];
    for (int i=0;i<30;i++){
        tmp[i]=-1;
    }
    for (int i =0;i<N_f;i++){
        if (f[i]==vert_idx){
            p_in = i%3;
            face_idx = i/3;
            int v1 = f[face_idx*3+(p_in+1)%3];
            int v2 = f[face_idx*3+(p_in+2)%3];
            for (int j=0;j<num_neighbor;j++){
                if (tmp[j*2]==-1){
                    tmp[j*2]=v1;
                    tmp[j*2+1]=v2;
                    break;
                }
            }
        }
    }
    // sort neighbors
    adj[vert_idx*num_neighbor] = tmp[0];
    adj[vert_idx*num_neighbor+1] = tmp[1];
    for (int idx=2;idx<num_neighbor;idx++){
        for (int i=0;i<num_neighbor;i++){
            if (tmp[i*2]==adj[vert_idx*num_neighbor+idx-1]){
                //full chain of neighbors
                if (tmp[i*2+1]==tmp[0]){
                    return;
                }
                adj[vert_idx*num_neighbor+idx]=tmp[i*2+1];
                break;
            }
            // no chain
            else if (tmp[i*2]==-1){
                return;
            }
        }
    }
    
}
__device__ int valence(int*adj, int face_idx,int num_neighbor){
    int valence = 0;
    for (int j=0;j<num_neighbor;j++){
        if(adj[face_idx*num_neighbor+j]==-1){
            break;
        }
        valence++;   
    }
    return valence;
}

__device__ bool regular(int* adj, int* face,int num_neighbor){
    for (int i=0;i<3;i++){
        if (valence(adj,face[i],num_neighbor)!=6){
            return false;
        }
    }
    return true;
}

__device__ void collect_regular(int* collected_patch, int* adj,int* face,int num_neighbor,int p_per_patch){
    int present_thread = blockIdx.x * blockDim.x + threadIdx.x;
    int adj_order[3];
    for (int i=0;i<3;i++){
        for (int j=0;j<6;j++){
            if (adj[face[i]*num_neighbor+j]==face[(i+1)%3]){
                adj_order[i] = j;
                break;
            }
        }
    }
    collected_patch[present_thread*p_per_patch] = adj[face[0]*num_neighbor+(adj_order[0]+4)%6];
    collected_patch[present_thread*p_per_patch+1] = adj[face[0]*num_neighbor+(adj_order[0]+3)%6]; 
    collected_patch[present_thread*p_per_patch+2] = adj[face[0]*num_neighbor+(adj_order[0]+5)%6];
    collected_patch[present_thread*p_per_patch+3] = face[0];
    collected_patch[present_thread*p_per_patch+4] = adj[face[0]*num_neighbor+(adj_order[0]+2)%6];
    collected_patch[present_thread*p_per_patch+5] = adj[face[1]*num_neighbor+(adj_order[1]+3)%6];
    collected_patch[present_thread*p_per_patch+6] = face[1];
    collected_patch[present_thread*p_per_patch+7] = face[2];
    collected_patch[present_thread*p_per_patch+8] = adj[face[2]*num_neighbor+(adj_order[2]+4)%6];
    collected_patch[present_thread*p_per_patch+9] = adj[face[1]*num_neighbor+(adj_order[1]+4)%6];
    collected_patch[present_thread*p_per_patch+10] = adj[face[2]*num_neighbor+(adj_order[2]+2)%6];
    collected_patch[present_thread*p_per_patch+11] = adj[face[2]*num_neighbor+(adj_order[2]+3)%6];
}
__device__ void collect_irregular(int* collected_patch, int* adj,int* face,int num_neighbor,int p_per_patch){
    int present_thread = blockIdx.x * blockDim.x + threadIdx.x;
    int adj_order[3];
    int face_ordered[3];
    int N;
    for (int i=0;i<3;i++){
        N = valence(adj,face[i],num_neighbor);
        if (N!=6){
            face_ordered[0] = face[i];
            face_ordered[1] = face[(i+1)%3];
            face_ordered[2] = face[(i+2)%3];
            break;
        }
    }
    for (int i=0;i<3;i++){
        for (int j=0;j<6;j++){
            if (adj[face_ordered[i]*num_neighbor+j]==face_ordered[(i+1)%3]){
                adj_order[i] = j;
                break;
            }
        }
    }
    collected_patch[present_thread*p_per_patch] = face_ordered[0];
    collected_patch[present_thread*p_per_patch+1] = face_ordered[1];
    for (int i=N;i>1;i--){
        collected_patch[present_thread*p_per_patch+i]=adj[face_ordered[0]*num_neighbor+(adj_order[0]+N-i+1)%N];
    }
    collected_patch[present_thread*p_per_patch+N+1] = adj[face_ordered[2]*num_neighbor+(adj_order[2]+2)%6];
    collected_patch[present_thread*p_per_patch+N+2] = adj[face_ordered[1]*num_neighbor+(adj_order[1]+4)%6];
    collected_patch[present_thread*p_per_patch+N+3] = adj[face_ordered[1]*num_neighbor+(adj_order[1]+3)%6];
    collected_patch[present_thread*p_per_patch+N+4] = adj[face_ordered[2]*num_neighbor+(adj_order[2]+3)%6];
    collected_patch[present_thread*p_per_patch+N+5] = adj[face_ordered[2]*num_neighbor+(adj_order[2]+4)%6];
}

__global__ void collect_patch(int* f, int N_f, int* collected_patch,int *adj,int num_neighbor,int p_per_patch){
    int present_thread = blockIdx.x * blockDim.x + threadIdx.x;
    if (present_thread>N_f/3-1)
        return;
    int face_idx = present_thread;
    int face[3];
    for (int i=0;i<3;i++){
        face[i] = f[face_idx*3+i];
    }
    
    if (regular(adj,face,num_neighbor)){
        collect_regular(collected_patch,adj,face,num_neighbor,p_per_patch);
    }
    else{
        collect_irregular(collected_patch,adj,face,num_neighbor,p_per_patch);
    }

}

void write(const std::string path,thrust::host_vector<int>& adj,int num_neighbor){
    std::ofstream fout(path);
    for (int i=0;i<adj.size()/num_neighbor;i++){
        for (int j=0; j<num_neighbor;j++){
            if (j==num_neighbor-1){
                fout<<adj[i*num_neighbor+j]<<"\n";
            }
            else{
                fout<<adj[i*num_neighbor+j]<<" ";
            }
        }
    }
}
void read_obj(const std::string path,thrust::host_vector<float>& vertices,thrust::host_vector<int>& faces){
    std::ifstream file(path);
    std::string line;
    std::string word;
    while (std::getline(file,line)){
        std::stringstream str(line);
        std::getline(str,word,' ');
        if (word[0] == 'v'){
            for (int i=0;i<3;i++){
                std::getline(str,word,' ');
                vertices.push_back(std::stof(word));
            }
        }
        else if (word[0] == 'f'){
            for (int i=0;i<3;i++){
                std::getline(str,word,' ');
                faces.push_back(std::stoi(word)-1);
            }
        }
    } 
}
int main(){
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    thrust::host_vector<float> vertices;
    thrust::host_vector<int> faces;
    int num_neighbor = 12;
    read_obj("./extracted_mesh_puppet.obj",vertices,faces);
    thrust::device_vector<float> d_vertices;
    thrust::device_vector<int> d_faces;
    thrust::device_vector<int> d_adj(vertices.size()/3*num_neighbor);
    thrust::device_vector<int> d_collected_patch(faces.size()/3*(num_neighbor+6));
    thrust::host_vector<int> adj(vertices.size()/3*num_neighbor);
    thrust::host_vector<int> patch(faces.size()/3*(num_neighbor+6));
    thrust::fill(d_adj.begin(), d_adj.end(), -1);
    thrust::fill(d_collected_patch.begin(), d_collected_patch.end(), -1);
    d_vertices = vertices;
    d_faces = faces;
    float* p_vertices = thrust::raw_pointer_cast(d_vertices.data());
    int* p_faces = thrust::raw_pointer_cast(d_faces.data());
    int* p_adj = thrust::raw_pointer_cast(d_adj.data());
    int* p_collected_patch = thrust::raw_pointer_cast(d_collected_patch.data());
    int thread_num = 512;
    int total_thread_num = vertices.size()/3;
    
    int block_num = total_thread_num/thread_num +1;
    
    
    sorted_adjacency_list<<< block_num,thread_num >>> (p_faces,d_faces.size(),p_adj,vertices.size()/3,num_neighbor);
    hipDeviceSynchronize();
    hipEventRecord(start);
    total_thread_num = faces.size()/3;
    std::cout<<total_thread_num<<std::endl;
    block_num = total_thread_num/thread_num +1;
    collect_patch<<< block_num,thread_num >>> (p_faces,d_faces.size(),p_collected_patch,p_adj,num_neighbor,num_neighbor+6);
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout<<milliseconds<<std::endl;
    adj = d_adj;
    patch = d_collected_patch;
    write("./adj.txt",adj,num_neighbor);
    write("./patch.txt",patch,num_neighbor+6);
    //hipDeviceReset();
    return 0;

}