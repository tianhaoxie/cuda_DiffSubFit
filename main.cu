#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <vector>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <stdio.h>
#include <sstream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <chrono>
#include "src/eigenStructure.h"
#include "src/adj_list.h"
#include "src/collect_points.h"


__global__ void evaluate(float** eigenValues,float** eigenVectors,float** Phi){
    for (int i=0;i<9;i++){
        printf("%f ", eigenValues[0][i]);
    }
}

void write(const std::string path,thrust::host_vector<int>& adj,int num_neighbor){
    std::ofstream fout(path);
    for (int i=0;i<adj.size()/num_neighbor;i++){
        for (int j=0; j<num_neighbor;j++){
            if (j==num_neighbor-1){
                fout<<adj[i*num_neighbor+j]<<"\n";
            }
            else{
                fout<<adj[i*num_neighbor+j]<<" ";
            }
        }
    }
}
void read_obj(const std::string path,thrust::host_vector<float>& vertices,thrust::host_vector<int>& faces){
    std::ifstream file(path);
    std::string line;
    std::string word;
    while (std::getline(file,line)){
        std::stringstream str(line);
        std::getline(str,word,' ');
        if (word[0] == 'v'){
            for (int i=0;i<3;i++){
                std::getline(str,word,' ');
                vertices.push_back(std::stof(word));
            }
        }
        else if (word[0] == 'f'){
            for (int i=0;i<3;i++){
                std::getline(str,word,' ');
                faces.push_back(std::stoi(word)-1);
            }
        }
    } 
}

int main(){
    hipEvent_t start, stop;
    hipEventCreate(&start);    hipEventCreate(&stop);
    eigenStructure* es = new eigenStructure();
    float** p_eigenValues;
    float** p_eigenVectors;
    float** p_Phi;
    thrust::host_vector<float> vertices;
    thrust::host_vector<int> faces;
    int num_neighbor = 12;
    read_obj("../extracted_mesh_puppet.obj",vertices,faces);
    thrust::device_vector<float> d_vertices;
    thrust::device_vector<int> d_faces;
    thrust::device_vector<int> d_adj(vertices.size()/3*num_neighbor);
    thrust::device_vector<int> d_collected_patch(faces.size()/3*(num_neighbor+6));
    thrust::host_vector<int> adj(vertices.size()/3*num_neighbor);
    thrust::host_vector<int> patch(faces.size()/3*(num_neighbor+6));
    thrust::fill(d_adj.begin(), d_adj.end(), -1);
    thrust::fill(d_collected_patch.begin(), d_collected_patch.end(), -1);
    d_vertices = vertices;
    d_faces = faces;
    float* p_vertices = thrust::raw_pointer_cast(d_vertices.data());
    int* p_faces = thrust::raw_pointer_cast(d_faces.data());
    int* p_adj = thrust::raw_pointer_cast(d_adj.data());
    int* p_collected_patch = thrust::raw_pointer_cast(d_collected_patch.data());
    hipHostAlloc((void**)&p_eigenValues, (num_neighbor-3)*sizeof(float*),hipHostMallocMapped);
    hipHostAlloc((void**)&p_eigenVectors, (num_neighbor-3)*sizeof(float*),hipHostMallocMapped);
    hipHostAlloc((void**)&p_Phi, (num_neighbor-3)*sizeof(float*),hipHostMallocMapped);
    es->to_device(p_eigenValues,p_eigenVectors,p_Phi);
    //evaluate<<<1,1>>> (p_eigenValues,p_eigenVectors,p_Phi);
    //hipDeviceSynchronize();
    
    int thread_num = 512;
    int total_thread_num = vertices.size()/3;
    int block_num = total_thread_num/thread_num +1;
    
    
    sorted_adjacency_list<<< block_num,thread_num >>> (p_faces,d_faces.size(),p_adj,vertices.size()/3,num_neighbor);
    hipDeviceSynchronize();
    hipEventRecord(start);
    total_thread_num = faces.size()/3;
    std::cout<<total_thread_num<<std::endl;
    block_num = total_thread_num/thread_num +1;
    collect_patch<<< block_num,thread_num >>> (p_faces,d_faces.size(),p_collected_patch,p_adj,num_neighbor,num_neighbor+6);
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout<<milliseconds<<std::endl;
    adj = d_adj;
    patch = d_collected_patch;
    write("./adj.txt",adj,num_neighbor);
    write("./patch.txt",patch,num_neighbor+6);
    //hipDeviceReset();
    
    return 0;

}